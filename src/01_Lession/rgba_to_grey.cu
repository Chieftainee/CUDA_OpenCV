#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "opencv2/core/cuda.hpp"
#include "opencv2/core/cuda_stream_accessor.hpp"

#define CHECK(call)											   				\
{															   				\
	const hipError_t error = call;							   				\
	if (error != hipSuccess)								   				\
	{														   				\
		fprintf(stderr, "Error: ##########################\n");				\
		fprintf(stderr, "FILE: %s\n", __FILE__);               				\
		fprintf(stderr, "LINE: %d\n", __LINE__);               				\
		fprintf(stderr, "CUDA Code: %d, reason: %s\n",		   				\
			error, hipGetErrorString(error));				   				\
		fprintf(stderr, "##########################: Error\n");				\
		exit(1);											   				\
	}														   				\
}

__global__ 
void kernel(const cv::cuda::PtrStepSz<uchar4> src, cv::cuda::PtrStep<uchar1> dst)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if(ix < src.cols && iy < src.rows)
    {
        uchar4 color = src(iy, ix);
        dst(iy, ix) = make_uchar1(0.299f * color.x + 0.587f * color.y + 0.114f * color.z);
    }
}

// void gpu_rgba_to_greyscale(const cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst, cv::cuda::Stream& stream)
void gpu_rgba_to_greyscale(const cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst)
{
    // kernel函数配置
    dim3 block(16, 16);
    dim3 grid((src.cols + block.x - 1) / block.x, (src.rows + block.y - 1) / block.y);
    printf("cuda kernel grid  = (%d, %d, %d)\n", grid.x, grid.y, grid.z);
    printf("cuda kernel block = (%d, %d, %d)\n", block.x, block.y, block.z);

    dst.create(src.size(), CV_8UC1);
    kernel<<<grid, block>>>(src, dst);
    CHECK(hipGetLastError());

    // Class StreamAccessor that enables getting hipStream_t from cuda::Stream
    // hipStream_t s = cv::cuda::StreamAccessor::getStream(stream);
    hipDeviceSynchronize();
}